#include "hip/hip_runtime.h"

#include <stdio.h>
#include <math.h>

__global__ void mykernel(float * A,float * B)
{
	int bng = blockIdx.x; //block no. in grid
	int ntpd = blockDim.x; //no.of threads per block
	int tnb = threadIdx.x; //thread no. in block or local thread id
	int gtid = bng * ntpd + tnb;  
	B[gtid] = sinf(A[gtid]);
}

int main()
{
	float* A, *B;
	float *d_A, *d_B;
	int blks,thrds;
	printf("Enter no. of blocks in 1D grid: ");
	scanf("%d",&blks);
	printf("Enter no. of threads in 1D block: ");
	scanf("%d",&thrds);
	int  N = blks*thrds;
	int S = N * sizeof(float);
	A = (float*)malloc(S);
	B = (float*)malloc(S);

    	hipMalloc((void**)&d_A , S);
	hipMalloc((void**)&d_B , S);

	printf("Enter %d angles (in degrees) in A: ",N);
	for(int i=0; i<N; i++)
	{
		scanf("%f",&A[i]);
		A[i] *= 22.0/(7.0*180); //convert to radian
	}

	hipMemcpy(d_A,A,S,hipMemcpyHostToDevice);

	mykernel<<<blks,thrds>>>(d_A,d_B);

	hipMemcpy(B,d_B,S,hipMemcpyDeviceToHost);

	printf("Result: ");
	for(int i=0; i<N; i++)
	{
		printf("%.2f ",B[i]);
	}
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	free(A);
	free(B);
	return 0;
}
