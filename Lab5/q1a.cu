//N blocks 1 thread each i.e grid size is N (corrected question)
#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void mykernel(int * A,int * B,int * C)
{
	int gtid = blockIdx.x;
	C[gtid] = A[gtid] + B[gtid];
}

int main()
{
	int* A, *B, *C;
	int *d_A, *d_B, *d_C;
	printf("Enter no. of elements: ");
	int  N;
	scanf("%d",&N);
	int S = N * sizeof(int);
	A = (int*)malloc(S);
	B = (int*)malloc(S);
	C = (int*)malloc(S);
	hipMalloc((void**)&d_A , S);
	hipMalloc((void**)&d_B , S);
	hipMalloc((void**)&d_C , S);

	printf("Enter elements in A: ");
	for(int i=0; i<N; i++)
	{
		scanf("%d",&A[i]);
	}

	printf("Enter elements in B: ");
	for(int i=0; i<N; i++)
	{
		scanf("%d",&B[i]);
	}
	hipMemcpy(d_A,A,S,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,S,hipMemcpyHostToDevice);
	mykernel<<<N,1>>>(d_A,d_B,d_C);

	hipMemcpy(C,d_C,S,hipMemcpyDeviceToHost);

	printf("Result: ");
	for(int i=0; i<N; i++)
	{
		printf("%d ",C[i]);
	}
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(A);
	free(B);
	free(C);
	return 0;
}