#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void mykernel(int * A,int * B,int * C,int * N)
{
	int bng = blockIdx.x; //block no. in grid
	int ntpd = blockDim.x; //no.of threads per block
	int tnb = threadIdx.x; //thread no. in block or local thread id
	int gtid = bng * ntpd + tnb;  
	if(gtid < *N)
		C[gtid] = A[gtid] + B[gtid];
}

int main()
{
	int* A, *B, *C;
	int *d_A, *d_B, *d_C;
	printf("Enter no. of elements: ");
	int  N;
	int *d_N;
	scanf("%d",&N);
	int S = N * sizeof(int);
	A = (int*)malloc(S);
	B = (int*)malloc(S);
	C = (int*)malloc(S);

    hipMalloc((void**)&d_A , S);
	hipMalloc((void**)&d_B , S);
	hipMalloc((void**)&d_C , S);
	hipMalloc((void**)&d_N , sizeof(int)); //no. of threads

	printf("Enter elements in A: ");
	for(int i=0; i<N; i++)
	{
		scanf("%d",&A[i]);
	}

	printf("Enter elements in B: ");
	for(int i=0; i<N; i++)
	{
		scanf("%d",&B[i]);
	}

	hipMemcpy(d_A,A,S,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,S,hipMemcpyHostToDevice);
	hipMemcpy(d_N,&N,sizeof(int),hipMemcpyHostToDevice);

	mykernel<<<ceil(N/256.0),256>>>(d_A,d_B,d_C,d_N);

	hipMemcpy(C,d_C,S,hipMemcpyDeviceToHost);

	printf("Result: ");
	for(int i=0; i<N; i++)
	{
		printf("%d ",C[i]);
	}
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(A);
	free(B);
	free(C);
	return 0;
}