#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void MatrixMul(int * A,int* B,int*C, int n, int m, int q)
{
    int row = threadIdx.y +blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    if(row < m && col < q)
    {
        int sum = 0;
        for(int k=0; k<n; k++)
            sum += A[row * n + k] * B[k * q + col];

        C[row * q + col] = sum;
    }
}


int main()
{
    printf("Enter matrix dimensions of A(m and n): ");
    int m,n;
    scanf("%d %d",&m,&n);
    int A[m*n];
    printf("Enter matrix dimensions of B(p and q): ");
    int p,q;
    scanf("%d %d",&p,&q);
    if(n!=p)
    {
        printf("Matrix multiplication not possible\n");
        exit(-1);
    }
    int B[p*q];

    int k=0;
    printf("Enter matrix A: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&A[k++]); //directly reading as 1D array
        }    
    }

    k=0;
    printf("Enter matrix B: \n");
    for(int i=0;i<p;i++)
    {
        for(int j=0;j<q;j++)
        {
            scanf("%d",&B[k++]);
        }   
    }

    int C[m*q];
    int *d_A,*d_B,*d_C;
    hipMalloc((void**)&d_A,m*n*sizeof(int));
    hipMalloc((void**)&d_B,p*q*sizeof(int));
    hipMalloc((void**)&d_C,m*q*sizeof(int));

    hipMemcpy(d_A,A,m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,p*q*sizeof(int),hipMemcpyHostToDevice);



    dim3 blk(2,2,1);
    dim3 grid(ceil(q/2.0),ceil(m/2.0),1);


    MatrixMul<<<grid,blk>>>(d_A,d_B,d_C,n,m,q);

    hipMemcpy(C,d_C,m*q*sizeof(int),hipMemcpyDeviceToHost);

    k=0;
    printf("Final matrix C: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<q;j++)
        {
            printf("%d ",C[k++]);
        }
        printf("\n");   
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}