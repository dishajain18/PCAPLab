#include "hip/hip_runtime.h"

#include <stdio.h>

__device__ int get_gtid()
{
    int bng = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int ntpb = blockDim.x * blockDim.y * blockDim.z;
    int tnb = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int gtid = bng * ntpb + tnb;
    return gtid;
}

__global__ void odd_kernel(int *A,int m)
{
    int gtid = get_gtid();
    if(gtid < m && gtid % 2 == 1)
    {
        int temp;
        if(gtid + 1 <= m-1 && A[gtid] > A[gtid+1])
        {
            temp = A[gtid];
            A[gtid] = A[gtid + 1];                
            A[gtid + 1] = temp;
        }
    }
}


__global__ void even_kernel(int *A,int m)
{
    int gtid = get_gtid();
    if(gtid < m && gtid % 2 == 0)
    {
        int temp;
        if(gtid + 1 <= m-1 && A[gtid] > A[gtid+1])
        {
            temp = A[gtid];
            A[gtid] = A[gtid + 1];                
            A[gtid + 1] = temp;
        }
    }    
}



int main()
{
    int n;
    printf("Enter no. of elements: ");
    scanf("%d",&n);

    int size =  n * sizeof(int);
    int A[n];
    int *d_A;

    printf("Enter elements in A: ");
    for(int i=0; i< n; i++)
        scanf("%d",&A[i]);

    hipMalloc((void**)&d_A , size);

    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
    dim3 blk(1,3,2); //i.e 6 threads per block

    for(int i=0; i <= n/2; i++) // <= is kinda like ceil value in case of odd n
    {
        odd_kernel<<<ceil(n/6.0),blk>>>(d_A,n);
        even_kernel<<<ceil(n/6.0),blk>>>(d_A,n);
    }

    hipMemcpy(A,d_A,size,hipMemcpyDeviceToHost);

    printf("Sorted array: ");
	for(int i=0; i<n; i++)
	{
		printf("%d ",A[i]);
	}
	printf("\n");

	hipFree(d_A);
}
