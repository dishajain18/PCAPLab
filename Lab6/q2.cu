#include "hip/hip_runtime.h"

#include <stdio.h>

__device__ int get_gtid()
{
    int bng = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int ntpb = blockDim.x * blockDim.y * blockDim.z;
    int tnb = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int gtid = bng * ntpb + tnb;
    return gtid;
}

__global__ void ssort(int*A,int*B,int n)
{
    int gtid = get_gtid();
    if(gtid < n)
    {
        int pos = 0;
        int i;
        int data = A[gtid];
        for(i=0;i<n;i++)
        {
            if(A[i] < data || (A[i]==data && i < gtid))
                pos++;
        }
        B[pos]=data;
    }
}

int main()
{
    int n;
    printf("Enter no. of elements: ");
    scanf("%d",&n);

    int size =  n * sizeof(int);
    int A[n],B[n];
    int *d_A,*d_B;

    printf("Enter elements in A: ");
    for(int i=0; i< n; i++)
        scanf("%d",&A[i]);

    hipMalloc((void**)&d_A , size);
    hipMalloc((void**)&d_B , size);

    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
    dim3 blk(3,1,2); //i.e 6 threads per block

    ssort<<<ceil(n/6.0),blk>>>(d_A,d_B,n);

    hipMemcpy(B,d_B,size,hipMemcpyDeviceToHost);

    printf("Sorted array: ");
	for(int i=0; i<n; i++)
	{
		printf("%d ",B[i]);
	}
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	return 0;

}
