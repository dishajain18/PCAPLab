#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "stb_image.h"
#include "stb_image_write.h"

#define BLOCK_SIZE 16  // Define CUDA block size

__device__ int device_min(int a, int b) {
    return (a < b) ? a : b;
}

__device__ int device_max(int a, int b) {
    return (a > b) ? a : b;
}

__global__ void sobelEdgeDetection(unsigned char* d_input, unsigned char* d_output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;  // Prevent out-of-bounds memory access

    int Gx[3][3] = {{-1, 0, 1},
                    {-2, 0, 2},
                    {-1, 0, 1}};

    int Gy[3][3] = {{-1, -2, -1},
                    { 0,  0,  0},
                    { 1,  2,  1}};
    
    int sumX = 0, sumY = 0;
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            int px = device_min(device_max(x + i, 0), width - 1);
            int py = device_min(device_max(y + j, 0), height - 1);
            int pixel = d_input[py * width + px];
            sumX += pixel * Gx[i + 1][j + 1];
            sumY += pixel * Gy[i + 1][j + 1];
        }
    }

    int edgeValue = device_min(device_max(abs(sumX) + abs(sumY), 0), 255);
    d_output[y * width + x] = edgeValue;
}

void processImage(const char* inputFile, const char* outputFile) {
    int width, height, channels;
    unsigned char* h_input = stbi_load(inputFile, &width, &height, &channels, 1);
    if (!h_input) {
        printf("Error loading image!\n");
        return;
    }
    
    unsigned char *d_input, *d_output;
    hipMalloc((void**)&d_input, width * height);
    hipMalloc((void**)&d_output, width * height);
    hipMemcpy(d_input, h_input, width * height, hipMemcpyHostToDevice);
    
    dim3 grid((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 blk(BLOCK_SIZE, BLOCK_SIZE);
    
    sobelEdgeDetection<<<grid, blk>>>(d_input, d_output, width, height, channels);
    hipDeviceSynchronize();
    
    unsigned char* h_output = (unsigned char*)malloc(width * height);
    hipMemcpy(h_output, d_output, width * height, hipMemcpyDeviceToHost);
    
    stbi_write_jpg(outputFile, width, height, 1, h_output, 100);
    
    hipFree(d_input);
    hipFree(d_output);
    stbi_image_free(h_input);
    free(h_output);
}

int main(int argc, char** argv) {
    if (argc != 3) {
        printf("Usage: %s <input image> <output image>\n", argv[0]);
        return -1;
    }
    processImage(argv[1], argv[2]);
    return 0;
}
