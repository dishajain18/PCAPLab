#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void kernelele(int * A,int * B,int * C)
{
    int col = threadIdx.x;
    int row = threadIdx.y;
    int n = blockDim.x; // could have passed as an argument to function also but why not

    C[row*n+col] = A[row*n+col] + B[row*n+col];
}

int main()
{
    printf("Enter matrix dimensions(m and n): ");
    int m,n;
    scanf("%d %d",&m,&n);
    int A[m*n],B[m*n],C[m*n];
    int k=0;
    printf("Enter matrix A: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&A[k++]); //directly reading as 1D array
        }    
    }

    k=0;
    printf("Enter matrix B: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&B[k++]);
        }   
    }

    int *d_A,*d_B,*d_C;
    hipMalloc((void**)&d_A,m*n*sizeof(int));
    hipMalloc((void**)&d_B,m*n*sizeof(int));
    hipMalloc((void**)&d_C,m*n*sizeof(int));

    hipMemcpy(d_A,A,m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,m*n*sizeof(int),hipMemcpyHostToDevice);

    dim3 blk(n,m,1);
    kernelele<<<1,blk>>>(d_A,d_B,d_C);

    hipMemcpy(C,d_C,m*n*sizeof(int),hipMemcpyDeviceToHost);

    k=0;
    printf("Final matrix C: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            printf("%d ",C[k++]);
        }
        printf("\n");   
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}