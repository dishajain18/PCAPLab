#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void kernelcol(int * A,int * B,int * C,int m,int n)
{
    int col = threadIdx.x;
    int q = blockDim.x;

    for(int i=0;i<m;i++)
    {
        int sum = 0;
        for(int j=0;j<n;j++)
            sum += A[i*n+j] * B[j*q+col];
        C[i*q+col] = sum;
    }
        
}

int main()
{
    printf("Enter matrix dimensions of A(m and n): ");
    int m,n;
    scanf("%d %d",&m,&n);
    int A[m*n];
    int k=0;
    printf("Enter matrix A: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&A[k++]); //directly reading as 1D array
        }    
    }

    printf("Enter matrix dimensions of B(p and q): ");
    int p,q;
    scanf("%d %d",&p,&q);
    if(n!=p)
    {
        printf("Matrix multiplication not possible\n");
        exit(-1);
    }
    int B[p*q];
    k=0;
    printf("Enter matrix B: \n");
    for(int i=0;i<p;i++)
    {
        for(int j=0;j<q;j++)
        {
            scanf("%d",&B[k++]);
        }   
    }

    int C[m*q];
    int *d_A,*d_B,*d_C;
    hipMalloc((void**)&d_A,m*n*sizeof(int));
    hipMalloc((void**)&d_B,p*q*sizeof(int));
    hipMalloc((void**)&d_C,m*q*sizeof(int));

    hipMemcpy(d_A,A,m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,p*q*sizeof(int),hipMemcpyHostToDevice);

    kernelcol<<<1,q>>>(d_A,d_B,d_C,m,n);

    hipMemcpy(C,d_C,m*q*sizeof(int),hipMemcpyDeviceToHost);

    k=0;
    printf("Final matrix C: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<q;j++)
        {
            printf("%d ",C[k++]);
        }
        printf("\n");   
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}