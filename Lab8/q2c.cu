#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void kernelele(int * A,int * B,int * C,int n)
{
    int row = threadIdx.y;
    int col = threadIdx.x;
    int q = blockDim.x;
    int sum = 0;

    for(int k=0;k<n;k++)
            sum += A[row*n+k] * B[k*q+col];
    
    C[row*q+col] = sum;
        
}

int main()
{
    printf("Enter matrix dimensions of A(m and n): ");
    int m,n;
    scanf("%d %d",&m,&n);
    int A[m*n];
    printf("Enter matrix dimensions of B(p and q): ");
    int p,q;
    scanf("%d %d",&p,&q);
    if(n!=p)
    {
        printf("Matrix multiplication not possible\n");
        exit(-1);
    }
    int B[p*q];

    int k=0;
    printf("Enter matrix A: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&A[k++]); //directly reading as 1D array
        }    
    }

    k=0;
    printf("Enter matrix B: \n");
    for(int i=0;i<p;i++)
    {
        for(int j=0;j<q;j++)
        {
            scanf("%d",&B[k++]);
        }   
    }
    
    int C[m*q];
    int *d_A,*d_B,*d_C;
    hipMalloc((void**)&d_A,m*n*sizeof(int));
    hipMalloc((void**)&d_B,p*q*sizeof(int));
    hipMalloc((void**)&d_C,m*q*sizeof(int));

    hipMemcpy(d_A,A,m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,p*q*sizeof(int),hipMemcpyHostToDevice);

    dim3 blk(q,m,1);
    kernelele<<<1,blk>>>(d_A,d_B,d_C,n);

    hipMemcpy(C,d_C,m*q*sizeof(int),hipMemcpyDeviceToHost);

    k=0;
    printf("Final matrix C: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<q;j++)
        {
            printf("%d ",C[k++]);
        }
        printf("\n");   
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
