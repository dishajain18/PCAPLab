#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void kernelrow(int * A,int * B,int * C,int n)
{
    int row = threadIdx.x;

    for(int i=0;i<n;i++)
        C[row*n+i] = A[row*n+i] + B[row*n+i];
}

int main()
{
    printf("Enter matrix dimensions(m and n): ");
    int m,n;
    scanf("%d %d",&m,&n);
    int A[m*n],B[m*n],C[m*n];
    int k=0;
    printf("Enter matrix A: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&A[k++]); //directly reading as 1D array
        }    
    }

    k=0;
    printf("Enter matrix B: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&B[k++]);
        }   
    }

    int *d_A,*d_B,*d_C;
    hipMalloc((void**)&d_A,m*n*sizeof(int));
    hipMalloc((void**)&d_B,m*n*sizeof(int));
    hipMalloc((void**)&d_C,m*n*sizeof(int));

    hipMemcpy(d_A,A,m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,m*n*sizeof(int),hipMemcpyHostToDevice);

    kernelrow<<<1,m>>>(d_A,d_B,d_C,n);

    hipMemcpy(C,d_C,m*n*sizeof(int),hipMemcpyDeviceToHost);

    k=0;
    printf("Final matrix C: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            printf("%d ",C[k++]);
        }
        printf("\n");   
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}