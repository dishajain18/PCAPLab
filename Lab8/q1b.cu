#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void kernelcol(int * A,int * B,int * C,int m)
{
    int col = threadIdx.x;
    int n = blockDim.x; // could have passed as an argument to function also but why not

    for(int i=0;i<m;i++)
        C[i*n+col] = A[i*n+col] + B[i*n+col];
}

int main()
{
    printf("Enter matrix dimensions(m and n): ");
    int m,n;
    scanf("%d %d",&m,&n);
    int A[m*n],B[m*n],C[m*n];
    int k=0;
    printf("Enter matrix A: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&A[k++]); //directly reading as 1D array
        }    
    }

    k=0;
    printf("Enter matrix B: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&B[k++]);
        }   
    }

    int *d_A,*d_B,*d_C;
    hipMalloc((void**)&d_A,m*n*sizeof(int));
    hipMalloc((void**)&d_B,m*n*sizeof(int));
    hipMalloc((void**)&d_C,m*n*sizeof(int));

    hipMemcpy(d_A,A,m*n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,m*n*sizeof(int),hipMemcpyHostToDevice);

    kernelcol<<<1,n>>>(d_A,d_B,d_C,m);

    hipMemcpy(C,d_C,m*n*sizeof(int),hipMemcpyDeviceToHost);

    k=0;
    printf("Final matrix C: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            printf("%d ",C[k++]);
        }
        printf("\n");   
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}