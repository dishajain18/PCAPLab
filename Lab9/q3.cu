#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void kernelele(int * A,int * B)
{
    int col = threadIdx.x;
    int row = threadIdx.y;
    int n = blockDim.x; 
    int m = blockDim.y;
    if(row != 0 && row != m-1 && col != 0 && col != n-1)
    {
        int ele = A[row*n+col];
        int ones = 0;
        int pos = 1;
        if(ele==0)
         ones = 1;
        while(ele != 0)
        {
            int bit = ele % 2;
            bit = bit ^ 1; //XOR with 1 flips the bit
            ones += bit*pos;
            pos *= 10;
        }

        B[row*n+col]=ones;
    }

}

int main()
{
    printf("Enter matrix dimensions(m and n): ");
    int m,n;
    scanf("%d %d",&m,&n);

    int A[m*n],B[m*n];
    int k=0;
    printf("Enter matrix A: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&A[k++]); //directly reading as 1D array
        }  
    }

    int *d_A,*d_B;
    hipMalloc((void**)&d_A,m*n*sizeof(int));
    hipMalloc((void**)&d_B,m*n*sizeof(int));

    hipMemcpy(d_A,A,m*n*sizeof(int),hipMemcpyHostToDevice);

    dim3 blk(n,m,1);
    kernelele<<<1,blk>>>(d_A,d_B);

    hipMemcpy(B,d_B,m*n*sizeof(int),hipMemcpyDeviceToHost);

    k=0;
    printf("Final matrix: \n");
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            printf("%d ",B[k++]);
        }
        printf("\n");   
    }

    hipFree(d_A);
    hipFree(d_B);
    return 0;
}