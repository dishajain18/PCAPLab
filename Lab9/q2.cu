#include "hip/hip_runtime.h"

#include <stdio.h>

//for some reason pow(3,1) is coming 2, works fine for all other numbers so not using pow
__global__ void mykernel(int * A, int * B, int n)
{
    int row = threadIdx.x;
    for(int i=0; i<n;i++)
    {
        B[row*n+i] = 1;
        for(int j=1;j<=row+1;j++)
            B[row*n+i] *= A[row*n+i];
    }
}

int main()
{
    printf("Enter dimensions of input matrix: ");
    int m,n;
    scanf("%d %d",&m,&n);
    int A[m*n],B[m*n];
    printf("Enter the matrix: \n");
    int k=0;
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            scanf("%d",&A[k++]);
        }
    }

    int *d_A, *d_B;
    hipMalloc((void**)&d_A,k*sizeof(int));
    hipMalloc((void**)&d_B,k*sizeof(int));

    hipMemcpy(d_A,A,k*sizeof(int),hipMemcpyHostToDevice);

    mykernel<<<1,m>>>(d_A,d_B,n); //per thread one type of power i.e. thread 1 = same, thread 2 = square ...

    hipMemcpy(B,d_B,k*sizeof(int),hipMemcpyDeviceToHost);

    printf("Result: \n");

    k=0;
    for(int i=0;i<m;i++)
    {
        for(int j=0;j<n;j++)
        {
            printf("%d ",B[k++]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);

}
